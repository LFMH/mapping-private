#include "hip/hip_runtime.h"
#include <pcl/pcl_macros.h>

#include "boxlist_ray_intersection.h"
#include <thrust/sequence.h>
#include <iostream>
#include <pcl/cuda/time_cpu.h>
#include <thrust/device_ptr.h>

namespace realtime_perception 
{
  BoxListRayIntersection::BoxListRayIntersection (float3* min, float3* max, int numboxes, float3 origin, float focallength, int width, int height) 
    : min (min), max (max), numboxes (numboxes), origin (origin), focallength (focallength), width (width), height (height)
  {}

  template <typename Tuple> __inline__ __host__ __device__
    int BoxListRayIntersection::operator() (const Tuple &t) const
  {
    Intersection is = intersect_boxes (thrust::get<0>(t));
    if (is.tmin == 0)
    {
      return 0;
    }
    else
    {
      return is.boxid + 1;
    }
  }

  BoxListRayIntersection::Intersection BoxListRayIntersection::intersect_boxes (int pt_index) const
  {
    Intersection ret;
    float min_tmin = ret.tmin = 0;

    for (int i = 0; i < numboxes; i++)
    {
      float3 ray_direction;
      ray_direction.x = pt_index;
      ray_direction.y = pt_index;
      ray_direction.z = pt_index;
      // TODO: compute ray direction from index
      float tmin = hitbox (i, ray_direction);
      if (tmin < min_tmin)
      {
        min_tmin = tmin;
        ret.tmin = tmin;
        ret.boxid = i;
      }
    }
    return ret;
  }

  float BoxListRayIntersection::hitbox (int index, float3 ray_direction) const
  {
    float3 &m1 = min[index];
    float3 &m2 = max[index];

    float tymin, tymax, tzmin, tzmax, tmin, tmax; 
    float flag = 1.0; 
  
     if (ray_direction.x >= 0) 
     {
        tmin = (m1.x - origin.x) / ray_direction.x;
        tmax = (m2.x - origin.x) / ray_direction.x;
     }
     else 
     {
        tmin = (m2.x - origin.x) / ray_direction.x;
        tmax = (m1.x - origin.x) / ray_direction.x;
     }
     if (ray_direction.y >= 0) 
     {
        tymin = (m1.y - origin.y) / ray_direction.y; 
        tymax = (m2.y - origin.y) / ray_direction.y; 
     }
     else 
     {
        tymin = (m2.y - origin.y) / ray_direction.y; 
        tymax = (m1.y - origin.y) / ray_direction.y; 
     }
      
     if ((tmin > tymax) || (tymin > tmax)) flag = -1.0; 
     if (tymin > tmin) tmin = tymin; 
     if (tymax < tmax) tmax = tymax; 
       
     if (ray_direction.z >= 0) 
     {
        tzmin = (m1.z - origin.z) / ray_direction.z; 
        tzmax = (m2.z - origin.z) / ray_direction.z; 
     }
     else 
     {
        tzmin = (m2.z - origin.z) / ray_direction.z; 
        tzmax = (m1.z - origin.z) / ray_direction.z; 
     }
     if ((tmin > tzmax) || (tzmin > tmax)) flag = -1.0; 
     if (tzmin > tmin) tmin = tzmin; 
     if (tzmax < tmax) tmax = tzmax; 
       
     return tmin;
  }

  template <template <typename> class Storage> void
  BoxListRayIntersection::BoxListRayIntersection (float3* min, float3* max, int numboxes, float3 origin, float focallength, int width, int height) 
  URDF::compute (const boost::shared_ptr<openni_wrapper::DepthImage>& depth_image,
                             const boost::shared_ptr<openni_wrapper::Image>& rgb_image,
                             float constant,
                             typename PointCloudAOS<Storage>::Ptr &output,
                             bool downsample, int stride, int smoothing_nr_iterations, int smoothing_filter_size) 
  {
  thrust::transform (make_zip_iterator (make_tuple (counting_iterator<int>(0))), 
                     make_zip_iterator (make_tuple (counting_iterator<int>(0))) + output->width *output->height,
                     output->point.begin (),
                     BoxListRayIntersection (min, max, numboxes, origin, focallength, output->width, output->height));
  }

};

