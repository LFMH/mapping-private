#include "hip/hip_runtime.h"
#include <realtime_perception/urdf_filtering.h>
#include <thrust/count.h>
#include <thrust/extrema.h>

namespace realtime_perception 
{

pcl::cuda::PointXYZRGB
TransformationKernel::operator () (const pcl::cuda::PointXYZRGB &p)
{
  pcl::cuda::PointXYZRGB pt_out;
  pt_out.x = p.x*transformation[0] + p.y*transformation[4] + p.z*transformation[8] + transformation[12], 
  pt_out.y = p.x*transformation[1] + p.y*transformation[5] + p.z*transformation[9] + transformation[13], 
  pt_out.z = p.x*transformation[2] + p.y*transformation[6] + p.z*transformation[10] + transformation[14], 
  pt_out.rgb = p.rgb;
  return pt_out;
}

float4
RotationKernel::operator () (const float4 &p)
{
  return make_float4(
      p.x*transformation[0] + p.y*transformation[4] + p.z*transformation[8] , 
      p.x*transformation[1] + p.y*transformation[5] + p.z*transformation[9] , 
      p.x*transformation[2] + p.y*transformation[6] + p.z*transformation[10],
      p.w);
}

 //////////////////////////////////////////////////////////////////////////
template <typename Tuple> float
BackgroundSubtractionKernel::operator () (const Tuple &t)
{
  float zNear = 0.1f;
  float zFar = 100.0f;
  if (thrust::get<0>(t) == 0) // assume no_sample_value and shadow_value are also 0
    return 98;
    //return false;
  else
  {
    float depth_gl = thrust::get<1>(t);
  //  depth_gl = zNear + ( zFar - zNear ) * depth_gl;
    depth_gl = (zFar * zNear / (zNear - zFar)) / (depth_gl - zFar / (zFar - zNear));
    float depth_ni = ((float)thrust::get<0>(t)) * 0.001f;
    return (fabs(depth_ni - depth_gl));
    //return (fabs(depth_ni - depth_gl) < disp_thresh);
  }
}


template <template <typename> class Storage>
  void find_extrema (typename Storage<float>::type &array, float& min, float &max)
{
  thrust::pair <thrust::detail::normal_iterator<thrust::device_ptr<float> >,
                thrust::detail::normal_iterator<thrust::device_ptr<float> > >
  //thrust::pair<thrust::device_ptr<float> , thrust::device_ptr<float> > 
  result = thrust::minmax_element(array.begin(), array.end());

  min = *result.first;
  max = *result.second;
}



struct isTrue
{
  __host__ __device__
  bool operator()(const bool &x)
  {
    return x;
  }
};


template <template <typename> class Storage>
int BackgroundSubtraction::fromGLDepthImage (const boost::shared_ptr<openni_wrapper::DepthImage>& depth_image,
                                              typename pcl::cuda::StoragePointer<Storage,float>::type gl_depth,
                                              float constant, float disp_thresh,
                                              typename Storage<float>::type &output,
                                              bool downsample, int stride)
{
  // Copy the depth data on the card TODO: get this from pcl::io::DisparityToCloud::compute !
  typename Storage<float>::type depth (depth_image->getWidth() * depth_image->getHeight());
  unsigned short* depth_buffer = (unsigned short*)depth_image->getDepthMetaData ().Data ();
  thrust::copy (depth_buffer, (unsigned short*)(&depth_buffer[depth_image->getWidth() * depth_image->getHeight()]), depth.begin ());

  output.resize (depth_image->getWidth() * depth_image->getHeight());

  int image_size = depth_image->getWidth() * depth_image->getHeight ();
  thrust::transform (
      thrust::make_zip_iterator (thrust::make_tuple (depth.begin (), gl_depth)),
      thrust::make_zip_iterator (thrust::make_tuple (depth.begin (), gl_depth)) + image_size,
      output.begin (), 
      BackgroundSubtractionKernel (disp_thresh));

//  return thrust::count_if (output.begin (), output.end (), isTrue());
  return 0;
}

template <template <typename> class Storage>
void transformPoints (typename pcl::cuda::PointCloudAOS<Storage>::Ptr &out, const typename pcl::cuda::PointCloudAOS<Storage>::Ptr &in, double* transform)
{
  out.reset (new typename pcl::cuda::PointCloudAOS<Storage>);

  pcl::cuda::Host<double>::type transform_host (transform, transform + 16);
  typename Storage<double>::type transform_storage = transform_host;
  out->points.resize (in->points.size ());
  out->width = in->width;
  out->height = in->height;
  out->is_dense = in->is_dense;
  thrust::transform (in->points.begin (), in->points.end (), out->points.begin (), TransformationKernel (thrust::raw_pointer_cast(&transform_storage[0])));
}

template <template <typename> class Storage>
void transformNormals (boost::shared_ptr<typename Storage<float4>::type> &out, const boost::shared_ptr<typename Storage<float4>::type> &in, double* transform)
{
  out.reset (new typename Storage<float4>::type);

  pcl::cuda::Host<double>::type transform_host (transform, transform + 16);
  typename Storage<double>::type transform_storage = transform_host;
  out->resize (in->size ());
  thrust::transform (in->begin (), in->end (), out->begin (), RotationKernel (thrust::raw_pointer_cast(&transform_storage[0])));
}

//template int
//BackgroundSubtraction::fromGLDepthImage<pcl::cuda::Host> (const boost::shared_ptr<openni_wrapper::DepthImage>& depth_image,
//                                                          pcl::cuda::StoragePointer<pcl::cuda::Host,float>::type gl_depth,
//                                                          float constant, float disp_thresh,
//                                                          pcl::cuda::Host<bool>::type &output,
//                                                          bool downsample, int stride);
//
template int
BackgroundSubtraction::fromGLDepthImage<pcl::cuda::Device> (const boost::shared_ptr<openni_wrapper::DepthImage>& depth_image,
                                                            typename pcl::cuda::StoragePointer<pcl::cuda::Device,float>::type gl_depth,
                                                            float constant, float disp_thresh,
                                                            pcl::cuda::Device<float>::type &output,
                                                            bool downsample, int stride);
  

template void find_extrema<pcl::cuda::Device> (thrust::device_vector<float> &array, float& min, float &max);

template 
void transformPoints<pcl::cuda::Device> (typename pcl::cuda::PointCloudAOS<pcl::cuda::Device>::Ptr &out, const typename pcl::cuda::PointCloudAOS<pcl::cuda::Device>::Ptr &in, double* transform);

template 
void transformNormals<pcl::cuda::Device> (boost::shared_ptr<typename pcl::cuda::Device<float4>::type> &out, const boost::shared_ptr<typename pcl::cuda::Device<float4>::type> &in, double* transform);
 
  
} // end namespace
